#include "hip/hip_runtime.h"
﻿//=============================================================================================
// Path tracing program
//=============================================================================================
#define _CRT_SECURE_NO_WARNINGS
#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <list>
#include <chrono>
#include <queue> 


#include "HDRLoader/HDRLoader.h"
#include "copyToDevice/copyToDevice.h"
#include "math/math.cuh"
#include "constants/constants.cuh"


// Pseudo-random number in [0,1)
//__host__ __device__ float random() { return (float)rand() / RAND_MAX; }


struct Rand {
	__device__ static float random(hiprandState_t* state) {
		return hiprand_uniform(state);
	}
};


// Material class
struct Material {
	vec3 diffuseAlbedo;	// probability of diffuse reflection
	vec3 mirrorAlbedo;	// probability of mirror like reflection

	Material(vec3 _diffuseAlbedo, vec3 _mirrorAlbedo) {
		diffuseAlbedo = _diffuseAlbedo;
		mirrorAlbedo = _mirrorAlbedo;
	}
};

// sample direction with cosine distribution, returns the pdf
__device__ float SampleDiffuse(const vec3& N, const vec3& inDir, vec3& outDir, hiprandState_t* state) {
	vec3 T = cross(N, vec3(1.0f, 0.0f, 0.0f));	// Find a Cartesian frame T, B, N where T, B are in the plane
	if (T.length() < epsilon) T = cross(N, vec3(0.0f, 0.0f, 1.0f));
	T = T.normalize();
	vec3 B = cross(N, T);


	//float u = Rand::random(state);
	//float v = Rand::random(state);

	//float alpha = M_PI * 2.0f * u;
	//float beta = asinf(sqrtf(v));
	
	//outDir = (T * cosf(alpha) + B * sinf(alpha)) * sinf(beta) + N * cosf(beta);
	//return 1 / (2.0f * M_PI) * sinf(2.0f * beta);	// pdf

	float x, y, z;
	do {
		x = 2.0f * Rand::random(state) - 1.0f;    // random point in [-1,-1;1,1]
		y = 2.0f * Rand::random(state) - 1.0f;
	} while (x * x + y * y > 1.0f);  // reject if not in circle
	z = sqrtf(1.0f - x * x - y * y);  // project to hemisphere

	outDir = N * z + T * x + B * y;
	return z / M_PI;	// pdf
}

// sample direction with cosine distribution, returns the pdf
__device__ float SamplePhong(const vec3& N, const vec3& inDir, vec3& outDir, hiprandState_t* state) {
	vec3 R = inDir - N * dot(N, inDir) * 2.0f;
	vec3 T = cross(R, N);
	if (T.length() < epsilon) T = cross(R, vec3(0.0f, 0.0f, 1.0f));
	T = T.normalize();
	vec3 B = cross(R, T);

	float u = Rand::random(state);
	float v = Rand::random(state);

	float n = 4.0f;
	float alpha = PI * 2.0f * u;
	float beta = powf(acosf(1.0f-v), 1.0f/(n+1.0f));


	outDir = (T * cosf(alpha) + B * sinf(alpha)) * sinf(beta) + R * cosf(beta);
	return (1.0f / (2.0f * M_PI)) * (n+1.0f) * powf(cosf(beta), n) * sinf(beta);    // pdf
}

// sample direction of a Dirac delta distribution, returns the pdf
__device__ float SampleMirror(const vec3& N, const vec3& inDir, vec3& outDir) {
	outDir = inDir - N * dot(N, inDir) * 2.0f;
	return 1.0f;    // pdf
}

// Hit of ray tracing
struct Hit {
	float t;		// ray parameter
	vec3 position;	// position of the intersection
	vec3 normal;	// normal of the intersected surface
	Material* material;	// material of the intersected surface
	__host__ __device__ Hit() { t = -1; }
};

// The ray to be traced
struct Ray {
	vec3 start, dir;
	__host__ __device__ Ray(vec3 _start, vec3 _dir) { start = _start; dir = _dir.normalize(); }
};


class Intersectable {
protected:
	Material* material;
public:
	Intersectable() {}
	Intersectable(Material* mat) { 
		copyToDevice(material, *mat, "material");
	}
};

// Sphere
struct Sphere : public Intersectable {
	vec3 center;
	float radius;

	Sphere() : Intersectable() {}

	Sphere(const vec3& _center, float _radius, Material* mat1) : Intersectable(mat1) {
		center = _center;
		radius = _radius;
	}
	__host__ __device__ Hit intersect(const Ray& ray) {
		Hit hit;
		vec3 dist = ray.start - center;
		float a = dot(ray.dir, ray.dir);
		float b = dot(dist, ray.dir) * 2.0f;
		float c = dot(dist, dist) - radius * radius;
		float discr = b * b - 4.0f * a * c;
		if (discr < 0.0f) return hit;
		float sqrt_discr = sqrtf(discr);
		float t1 = (-b + sqrt_discr) / 2.0f / a;	// t1 >= t2 for sure
		float t2 = (-b - sqrt_discr) / 2.0f / a;
		if (t1 <= 0.0f) return hit;
		hit.t = (t2 > 0.0f) ? t2 : t1;
		hit.position = ray.start + ray.dir * hit.t;
		hit.normal = (hit.position - center) / radius;
		hit.material = material;
		if (dot(hit.normal, ray.dir) > 0.0f) hit.normal = hit.normal * (-1.0f); // flip the normal, we are inside the sphere
		return hit;
	}
};

// Plane
struct Plane : public Intersectable {
	vec3 point, normal;

	Plane() : Intersectable() {}

	Plane(const vec3& _point, const vec3& _normal, Material* mat) : Intersectable(mat) {
		point = _point;
		normal = _normal.normalize();
	}
	__host__ __device__ Hit intersect(const Ray& ray) {
		Hit hit;
		float NdotV = dot(normal, ray.dir);
		if (fabsf(NdotV) < epsilon) return hit;
		float t = dot(normal, point - ray.start) / NdotV;
		if (t < epsilon) return hit;
		hit.t = t;
		hit.position = ray.start + ray.dir * hit.t;
		hit.normal = normal;
		//if ((hit.position - point).length() > 1.5) return Hit();
		if (dot(hit.normal, ray.dir) > 0.0f) hit.normal = hit.normal * (-1.0f); // flip the normal, we are inside the sphere
		hit.material = material;
		return hit;
	}
};


//Ring
struct Ring : public Intersectable {
	vec3 position, normal;
	float radius, height;

	Ring() : Intersectable() {}

	Ring(const vec3& _point, const vec3& _normal, float _height, float _radius, Material* mat) : Intersectable(mat) {
		position = _point;
		normal = _normal.normalize();
		height = _height;
		radius = _radius;
	}
	__host__ __device__ Hit intersect(const Ray& ray) {
		Hit hit;

		vec3 distance = ray.start - position; //start-center

		float coreDotDistance = dot(normal, distance); //core°distance
		float coreDotDir = dot(normal, ray.dir); //core ° dir

		float c = dot(distance - normal * coreDotDistance, distance - normal * coreDotDistance) - radius * radius;
		float b = 2.0f * dot(distance - normal * coreDotDistance, ray.dir - normal * coreDotDir);
		float a = dot(ray.dir - normal * coreDotDir, ray.dir - normal * coreDotDir);

		float discr = (b * b) - (4.0f * a * c);
		if (discr < 0.0f) return hit; //no real solution

		float sqrt_discr = sqrtf(discr);
		float t1 = (-b - sqrt_discr) / 2.0f / a;
		float t2 = (-b + sqrt_discr) / 2.0f / a;

		float t1Height = coreDotDistance + t1 * coreDotDir;
		float t2Height = coreDotDistance + t2 * coreDotDir;

		if (t2 <= 0.0f) { //neither of the solutions are on the "good" side of the ray, no intersection
			return hit;
		}
		if (t1 > 0.0f) {
			if (t1Height > 0.0f && t1Height < height) {
				hit.t = t1;
				hit.position = ray.start + ray.dir * t1;
				hit.normal = (position + normal * t1Height - hit.position).normalize();
				if (dot(hit.normal, ray.dir) > 0.0f) hit.normal = hit.normal * (-1.0f); // flip the normal
				hit.material = material;
				return hit;
			}
		}
		if (t2Height > 0.0f && t2Height < height) {
			hit.t = t2; 
			hit.position = ray.start + ray.dir * t2;
			hit.normal = (position + normal * t2Height - hit.position).normalize();
			if (dot(hit.normal, ray.dir) > 0.0f) hit.normal = hit.normal * (-1.0f); // flip the normal
			hit.material = material;
			return hit;
		}

		return hit;
	}
};


//Mesh
struct Vertex {
	vec3 position, normal;
	__host__ __device__ Vertex(vec3 position, vec3 normal) {
		this->position = position;
		this->normal = normal;
	}
	__host__ __device__  Vertex() {
		position = vec3(0, 0, 0);
		normal = vec3(0, 0, 0);
	}
};

struct Triangle {
	Vertex a, b, c;

	Triangle(Vertex a, Vertex b, Vertex c, Material* mat) {
		this->a = a;
		this->b = b;
		this->c = c;
	}

	Triangle() {}

	__host__ __device__ Hit intersect(const Ray& ray) {
		Hit hit;

		// compute plane's normal
		vec3 ab = b.position - a.position;
		vec3 ac = c.position - a.position;
		// no need to normalize
		vec3 N = cross(ab, ac); // N 
		float area2 = N.length();

		// Step 1: finding P

		// check if ray and plane are parallel ?
		float NdotRayDirection = dot(N, ray.dir);
		if (fabsf(NdotRayDirection) < epsilon) // almost 0 
			return hit; // they are parallel so they don't intersect ! 

		float d = -dot(N, a.position);
		hit.t = -(dot(N, ray.start) + d) / NdotRayDirection;

		// check if the triangle is in behind the ray
		if (hit.t < 0.0f) return hit; // the triangle is behind 

		// compute the intersection point using equation 1
		vec3 P = ray.start + ray.dir * hit.t;

		// Step 2: inside-outside test
		vec3 C; // vector perpendicular to triangle's plane 

		// edge 0
		vec3 vpA = P - a.position;
		C = cross(ab, vpA);
		if (dot(N, C) < 0.0f) {
			hit.t = -1.0f;
			return hit;
		} // P is on the right side 

		// edge 1
		vec3 bc = c.position - b.position;
		vec3 vpB = P - b.position;
		C = cross(bc, vpB);
		if (dot(N, C) < 0.0f) {
			hit.t = -1.0f;
			return hit;
		} // P is on the right side

		// edge 2
		vec3 ca = a.position - c.position;
		vec3 vpC = P - c.position;
		C = cross(ca, vpC);
		if (dot(N, C) < 0.0f) {
			hit.t = -1.0f;
			return hit;
		}  // P is on the right side; 

		hit.position = ray.start + ray.dir * hit.t;
		
		vec3 v_a, v_c, v_h;

		float bary[3];
		v_a = a.position - b.position;
		v_c = c.position - b.position;
		v_h = hit.position - b.position;

		float daa = dot(v_a, v_a);
		float dac = dot(v_a, v_c);
		float dcc = dot(v_c, v_c);
		float denom = daa * dcc - dac * dac;

		float dha = dot(v_h, v_a);
		float dhc = dot(v_h, v_c);
		bary[0] = (dcc * dha - dac * dhc) / denom;
		bary[1] = (daa * dhc - dac * dha) / denom;
		bary[2] = 1.0f - bary[0] - bary[1];
		hit.normal = (a.normal * bary[0] + b.normal * bary[2] + c.normal * bary[1]).normalize();
		if (dot(hit.normal, ray.dir) > 0.0f) hit.normal = hit.normal * (-1.0f); // flip the normal
		return hit; // this ray hits the triangle
	}
};

struct Box {
	float minX, maxX;
	float minY, maxY;
	float minZ, maxZ;
	Box(float minX, float maxX, float minY, float maxY, float minZ, float maxZ) {
		this->minX = minX;
		this->maxX = maxX;
		this->minY = minY;
		this->maxY = maxY;
		this->minZ = minZ;
		this->maxZ = maxZ;
	}
	Box() {}

	void half(Box& boxLeft, Box& boxRight, Axis axis, float boxLeftPercent = 0.5f) {
		boxLeft.minX = minX;
		boxLeft.maxX = maxX;
		boxLeft.minY = minY;
		boxLeft.maxY = maxY;
		boxLeft.minZ = minZ;
		boxLeft.maxZ = maxZ;
		boxRight.minX = minX;
		boxRight.maxX = maxX;
		boxRight.minY = minY;
		boxRight.maxY = maxY;
		boxRight.minZ = minZ;
		boxRight.maxZ = maxZ;
		if (axis == Axis_X) {
			boxLeft.maxX = minX + (maxX - minX) * boxLeftPercent;
			boxRight.minX = boxLeft.maxX;
		}
		if (axis == Axis_Y) {
			boxLeft.maxY = minY + (maxY - minY) * boxLeftPercent;
			boxRight.minY = boxLeft.maxY;
		}
		if (axis == Axis_Z) {
			boxLeft.maxZ = minZ + (maxZ - minZ) * boxLeftPercent;
			boxRight.minZ = boxLeft.maxZ;
		}
	}

	__host__ __device__ bool isVertexInBox(vec3 position) {
		return(minX <= position.x &&
			position.x <= maxX + epsilon &&
			minY <= position.y + epsilon &&
			position.y <= maxY + epsilon &&
			minZ <= position.z + epsilon &&
			position.z <= maxZ + epsilon
			);
	}

	__host__ __device__ bool isTriangleInBox(Triangle* triangle) {
		return(isVertexInBox(triangle->a.position) ||
			isVertexInBox(triangle->b.position) ||
			isVertexInBox(triangle->c.position)
			);
	}

	__host__ __device__ Hit intersect(const Ray& ray) {
		vec3 potentialPoint[6];
		float t[6];
		t[0] = (minX - ray.start.x) / ray.dir.x;
		t[1] = (maxX - ray.start.x) / ray.dir.x;
		potentialPoint[0] = ray.start + ray.dir * t[0];
		potentialPoint[1] = ray.start + ray.dir * t[1];
		t[2] = (minY - ray.start.y) / ray.dir.y;
		t[3] = (maxY - ray.start.y) / ray.dir.y;
		potentialPoint[2] = ray.start + ray.dir * t[2];
		potentialPoint[3] = ray.start + ray.dir * t[3];
		t[4] = (minZ - ray.start.z) / ray.dir.z;
		t[5] = (maxZ - ray.start.z) / ray.dir.z;
		potentialPoint[4] = ray.start + ray.dir * t[4];
		potentialPoint[5] = ray.start + ray.dir * t[5];

		Hit bestHit;
		for (int i = 0; i < 6; i++) {
			if (t[i] > 0.0f && (bestHit.t < 0.0f || t[i] < bestHit.t)
				&& isVertexInBox(potentialPoint[i])) bestHit.t = t[i];
		}
		return bestHit;
	}

	__host__ __device__ float widthOnAxis(Axis axis) {
		if (axis == Axis_X) return maxX - minX;
		if (axis == Axis_Y) return maxY - minY;
		if (axis == Axis_Z) return maxZ - minZ;
	}

	__host__ __device__ bool inBoundsOnAxis(Axis axis, float t) {
		if (axis == Axis_X) {
			return (t > minX && t < maxX);
		}
		if (axis == Axis_Y) {
			return (t > minY && t < maxY);
		}
		if (axis == Axis_Z) {
			return (t > minZ && t < maxZ);
		}
	}

	__host__ __device__ float startingPointOnAxis(Axis axis) {
		if (axis == Axis_X) return minX;
		if (axis == Axis_Y) return minY;
		if (axis == Axis_Z) return minZ;
	}
};


struct Node {
	//CPU
	Node* left = nullptr;
	Node* right = nullptr;

	//shared
	Box box;
	Axis axis;

	//CUDA
	Triangle* device_triangles;
	int* device_triangles_size;

	int left_child_of_parent_idx = -1;
	int right_child_of_parent_idx = -1;
	int left_index = -1;
	int right_index = -1;


	__device__ __host__ bool isLeaf() {
		return left == nullptr;
	}

	void makeNodeTree(std::vector<Triangle*> triangles, int depth=0) {
		if (triangles.size() <= maxTriangleNum || depth > maxKdTreeHeight) {
			int size = triangles.size();
			copyToDevice(device_triangles_size, size, "device_triangles_size");
			copyToDevice(device_triangles, triangles, size, "device_triangles");
			return;
		}
		left = new Node();
		right = new Node();


		//get vertices
		int N_vertices = triangles.size() * 3;
		float* vertices = new float[N_vertices];
		for (int i = 0; i < triangles.size(); i++) {
			vertices[3 * i] = triangles[i]->a.position.axisCoordinate(axis);
			vertices[3 * i + 1] = triangles[i]->b.position.axisCoordinate(axis);
			vertices[3 * i + 2] = triangles[i]->c.position.axisCoordinate(axis);
		}

		//remove vertices out of box bounds
		int N_verces_in_bounds = 0;
		for (int i = 0; i < N_vertices; i++) {
			if (box.inBoundsOnAxis(axis, vertices[i])) N_verces_in_bounds++;
		}

		float* vertices_in_bounds = new float[N_verces_in_bounds];
		int j = 0;
		for (int i = 0; i < N_vertices; i++) {
			if (box.inBoundsOnAxis(axis, vertices[i])) {
				vertices_in_bounds[j++] = vertices[i];
			}
		}
		delete[] vertices;

		//sort verticies
		std::sort(vertices_in_bounds, vertices_in_bounds + N_verces_in_bounds, std::less<float>());


		//log2 look for optimal halfing point
		int idx_split_point = N_verces_in_bounds / 2;
		for (int i = 2; i <= triangleOptimumSearchMaxDepth + 1; i++) {
			//split
			float splitPoint = vertices_in_bounds[idx_split_point];
			box.half(left->box, right->box, axis,
				(splitPoint - box.startingPointOnAxis(axis)) / box.widthOnAxis(axis)
			);
			//count triangles in each
			int N_trinagles_left = 0, N_triangles_right = 0;
			for (auto triangle : triangles) {
				if (left->box.isTriangleInBox(triangle)) {
					N_trinagles_left++;
				}
				if (right->box.isTriangleInBox(triangle)) {
					N_triangles_right++;
				}
			}
			//decide next move
			if ((N_trinagles_left - N_triangles_right) < allowedtriangleDifference
				&& -allowedtriangleDifference < (N_trinagles_left - N_triangles_right)) {
				break;
			}
			else if (N_trinagles_left > N_triangles_right) {
				idx_split_point -= N_verces_in_bounds / (2 * i);
				if (idx_split_point < 0) {
					idx_split_point = 0;
					break;
				}
			}
			else {
				idx_split_point += N_verces_in_bounds / (2 * i);
				if (idx_split_point > N_verces_in_bounds - 1) {
					idx_split_point = N_verces_in_bounds - 1;
					break;
				}
			}
		}


		//half at selected point
		box.half(left->box, right->box, axis
			, (vertices_in_bounds[idx_split_point] - box.startingPointOnAxis(axis)) / box.widthOnAxis(axis)
		);

		delete[] vertices_in_bounds;
		/////////////////////////////////////////


		//add triangles
		std::vector<Triangle*> left_triangles;
		std::vector<Triangle*> right_triangles;
		for (auto triangle : triangles) {
			if (left->box.isTriangleInBox(triangle)) {
				left_triangles.push_back(triangle);
			}
			if (right->box.isTriangleInBox(triangle)) {
				right_triangles.push_back(triangle);
			}
		}
		//change axis for next time
		left->axis = nextAxis(axis);
		right->axis = nextAxis(axis);
		left->makeNodeTree(left_triangles, depth + 1);
		right->makeNodeTree(right_triangles, depth + 1);
	}

	int getSize() {
		if (isLeaf()) return 1;
		return 1 + left->getSize() + right->getSize();
	}
};


struct Mesh {
	Node* device_kdTree;
	int* device_kdTreeSize;

	Mesh(std::vector<Triangle*> triangles) {
		Node* kdTree = getKdTree(triangles);
		uploadKdTree(kdTree);
		for (auto triangle : triangles) {
			delete triangle;
		}
	}
	Node* getKdTree(std::vector<Triangle*> triangles) {
		Node* kdTree = new Node();

		kdTree->box.minX = triangles[0]->a.position.x;
		kdTree->box.maxX = triangles[0]->a.position.x;
		kdTree->box.minY = triangles[0]->a.position.y;
		kdTree->box.maxY = triangles[0]->a.position.y;
		kdTree->box.minZ = triangles[0]->a.position.z;
		kdTree->box.maxZ = triangles[0]->a.position.z;
		for (auto triangle : triangles) {
			processTriangleToBox(triangle, kdTree->box);
		}

		kdTree->axis = Axis_X;
		kdTree->makeNodeTree(triangles);

		return kdTree;
	}
	void processTriangleToBox(Triangle* triangle, Box& box) {
		processVertexToBox(triangle->a, box);
		processVertexToBox(triangle->b, box);
		processVertexToBox(triangle->c, box);
	}
	void processVertexToBox(Vertex vertex, Box& box) {
		if (vertex.position.x < box.minX) box.minX = vertex.position.x;
		if (vertex.position.x > box.maxX) box.maxX = vertex.position.x;
		if (vertex.position.y < box.minY) box.minY = vertex.position.y;
		if (vertex.position.y > box.maxY) box.maxY = vertex.position.y;
		if (vertex.position.z < box.minZ) box.minZ = vertex.position.z;
		if (vertex.position.z > box.maxZ) box.maxZ = vertex.position.z;
	}


	void uploadKdTree(Node* kdTree) {
		//save size
		int size = kdTree->getSize();
		copyToDevice(device_kdTreeSize, size, "device_kdTreeSize");

		Node *host_kdTree = new Node[size];

		std::queue<Node*> queue;
		queue.push(kdTree);
		int idx = 0;
		while (queue.size() > 0) {
			Node* currentNode = queue.front();
			queue.pop();

			while (!(currentNode->isLeaf())) {
				putNodeToIdx(currentNode, host_kdTree, idx);
				idx++;
				
				//next while
				queue.push(currentNode->left);
				currentNode = currentNode->right;
			}
			//leaf node
			putNodeToIdx(currentNode, host_kdTree, idx);
			idx++;
		}
		//upload to CUDA
		copyToDevice(device_kdTree, host_kdTree, size, "device_kdTree");
	}

	void putNodeToIdx(Node* currentNode, Node* array, int idx) {
		//go back to parent and update
		if (currentNode->left_child_of_parent_idx != -1) {
			array[currentNode->left_child_of_parent_idx]
				.left_index = idx;
		}
		else if (currentNode->right_child_of_parent_idx != -1) {
			array[currentNode->right_child_of_parent_idx]
				.right_index = idx;
		}

		if (!currentNode->isLeaf()) {
			//save for children where to update later
			currentNode->left->left_child_of_parent_idx = idx;
			currentNode->right->right_child_of_parent_idx = idx;
		}

		//add to array
		array[idx] = *currentNode;
	}

	__device__ Hit intersect(const Ray& ray) {
		if (device_kdTree[0].box.intersect(ray).t > 0) {

			//put the topnode into the stack
			int stack[maxKdTreeHeight];
			int stack_idx = 0;
			stack[stack_idx++] = 0;

			while (stack_idx > 0) {
				//get the next node
				Node currentNode = device_kdTree[stack[--stack_idx]];
				//tree traversing
				while (!(currentNode.isLeaf())) {
					Hit rightHit =
						device_kdTree[currentNode.right_index].box.intersect(ray);
					Hit leftHit = 
						device_kdTree[currentNode.left_index].box.intersect(ray);
					
					Hit smallerHit = (rightHit.t < leftHit.t) ?
						rightHit : leftHit;
					Hit biggerHit = (rightHit.t < leftHit.t) ?
						leftHit : rightHit;

					int smallerIdx = (rightHit.t < leftHit.t) ?
						currentNode.right_index : currentNode.left_index;
					int biggerIdx = (rightHit.t < leftHit.t) ?
						currentNode.left_index : currentNode.right_index;

					currentNode = (smallerHit.t > 0) ?
						device_kdTree[smallerIdx] : device_kdTree[biggerIdx];

					//if the ray hits both boxes
					if (smallerHit.t > 0 && biggerHit.t > 0) {
						stack[stack_idx++] = biggerIdx;
					}
				}
				//leaf node
				Hit bestHit;
				for (int i = 0; i < *currentNode.device_triangles_size; i++) {
					Hit hit = currentNode.device_triangles[i].intersect(ray); //  hit.t < 0 if no intersection
					if (hit.t > 0 && 
						(bestHit.t < 0 || hit.t < bestHit.t) 
						&& currentNode.box.isVertexInBox(ray.start + ray.dir * hit.t)
					) {
						bestHit = hit;
					}
				}
				if (bestHit.t > 0) return bestHit;
			}
			return Hit();
		}
		return Hit();
	}
};

//OBJ
constexpr auto POSITION = "v";
constexpr auto NORMAL = "vn";
constexpr auto FACE = "f";
constexpr auto POSITION_IDX = 0;
constexpr auto NORMAL_IDX = 2;
constexpr auto MAX_IDX = 2;
constexpr auto DEVIDER = '/';
constexpr auto EMPTY = ' ';
Mesh* readObjIntoMesh(std::string location, mat4 SRTmtx) {
	//read into these
	std::vector<vec3> positions;
	std::vector<vec3> normals;
	std::vector<int> position_idx;
	std::vector<int> normal_idx;

	//open file
	std::stringstream ss;
	std::ifstream in_file(location);
	std::string line = "";
	if (!in_file.is_open()) {
		throw "Error opening file at " + location;
	}
	//helper temps
	std::string prefix = "";
	vec3 temp;
	int i;

	//read all lines
	while (std::getline(in_file, line)) {
		//clear, read new line
		ss.clear();
		ss.str(line);
		//get prefix
		ss >> prefix;

		if (prefix == POSITION)
		{
			ss >> temp.x >> temp.y >> temp.z;
			positions.push_back(temp);
		}
		else if (prefix == NORMAL)
		{
			ss >> temp.x >> temp.y >> temp.z;
			normals.push_back(temp);
		}
		else if (prefix == FACE)
		{
			int counter = 0;
			while (ss >> i) {
				//read
				if (counter == POSITION_IDX) {
					position_idx.push_back(i - 1); //we index from 0
				}
				else if (counter == NORMAL_IDX) {
					normal_idx.push_back(i - 1); //we index from 0
				}
				//handle separators
				for (int i = 0; i < 3; i++) {
					if (ss.peek() == DEVIDER) {
						counter++;
						ss.ignore(1, DEVIDER);
					}
				}
				if (ss.peek() == EMPTY) {
					counter = 0;
					ss.ignore(1, EMPTY);
				}
			}

		}
	}

	std::vector<Triangle*> triangles;
	triangles.resize(position_idx.size() / 3);
	for (size_t i = 0; i < triangles.size(); i++) {
		int vertexAIdx = i * 3;
		int vertexBIdx = i * 3 + 1;
		int vertexCIdx = i * 3 + 2;
		vec3 posA = (vec4(positions[position_idx[vertexAIdx]], 1) * SRTmtx ).xyz();
		vec3 normA = (vec4(normals[normal_idx[vertexAIdx]], 0) * SRTmtx.inverse().transpose()).xyz().normalize();
		vec3 posB = (vec4(positions[position_idx[vertexBIdx]], 1) * SRTmtx).xyz();
		vec3 normB = (vec4(normals[normal_idx[vertexBIdx]], 0) * SRTmtx.inverse().transpose()).xyz().normalize();
		vec3 posC = (vec4(positions[position_idx[vertexCIdx]], 1) * SRTmtx).xyz();
		vec3 normC = (vec4(normals[normal_idx[vertexCIdx]], 0) * SRTmtx.inverse().transpose()).xyz().normalize();

		triangles[i] = new Triangle(
			Vertex(posA, normA),
			Vertex(posB, normB),
			Vertex(posC, normC),
			nullptr
		);
	}
	return new Mesh(triangles);
}


struct MeshObject : public Intersectable {
	Mesh* device_mesh;
	vec3 positon = vec3(0, 0, 0);
	vec3 scale = vec3(1, 1, 1);
	vec3 rotate = vec3(0, 0, 0);

	MeshObject() : Intersectable() {
		
	}

	MeshObject(std::string location, vec3 position, vec3 rotate, vec3 scale, Material* mat) : Intersectable(mat) {
		Mesh* mesh = readObjIntoMesh(location, SRTmtx(scale, rotate, position));
		this->positon = position;
		this->scale = scale;
		this->rotate = rotate;
		copyToDevice(device_mesh, *mesh, "device_mesh");
	}

	__device__ Hit intersect(const Ray& ray) {
		Hit bestHit = device_mesh->intersect(ray);
		bestHit.material = this->material;
		return bestHit;
	}
};

class EnvMap {
	HDRLoaderResult px;
	HDRLoaderResult py;
	HDRLoaderResult pz;
	HDRLoaderResult nx;
	HDRLoaderResult ny;
	HDRLoaderResult nz;


	HDRLoaderResult* device_px;
	HDRLoaderResult* device_py;
	HDRLoaderResult* device_pz;
	HDRLoaderResult* device_nx;
	HDRLoaderResult* device_ny;
	HDRLoaderResult* device_nz;

	void copyHDRToCuda(HDRLoaderResult &cpu_result, HDRLoaderResult* &device_result) {
		//change cols to CUDA cols
		float* device_cols;
		copyToDevice(device_cols, cpu_result.cols, cpu_result.size(), "HDR.cols");
		delete cpu_result.cols;
		cpu_result.cols = device_cols;

		//copy to CUDA
		copyToDevice(device_result, cpu_result, "HDR");
	}
public:
	EnvMap(std::string folderName) {
		HDRLoader::load((folderName + "px.hdr").c_str(), px);
		HDRLoader::load((folderName + "py.hdr").c_str(), py);
		HDRLoader::load((folderName + "pz.hdr").c_str(), pz);
		HDRLoader::load((folderName + "nx.hdr").c_str(), nx);
		HDRLoader::load((folderName + "ny.hdr").c_str(), ny);
		HDRLoader::load((folderName + "nz.hdr").c_str(), nz);

		//Upload to CUDA
		copyHDRToCuda(px, device_px);
		copyHDRToCuda(py, device_py);
		copyHDRToCuda(pz, device_pz);
		copyHDRToCuda(nx, device_nx);
		copyHDRToCuda(ny, device_ny);
		copyHDRToCuda(nz, device_nz);
	}
	__host__ __device__ vec3 getPixelColor(vec3 ray) {
		HDRLoaderResult* hit = device_px;
		if ((abs(ray.x) >= abs(ray.y)) && (abs(ray.x) >= abs(ray.z))) {
			if (ray.x > 0.0f) hit = device_px;
			else hit = device_nx;

			int pos = ((int)((1.0f - ((ray.y / abs(ray.x) + 1.0f) / 2.0f)) * hit->height)) * 3 * hit->width
				+ ((int)((1.0f - ((ray.z / ray.x + 1.0f) / 2.0f)) * hit->width)) * 3;
			if (pos > hit->height * hit->width * 3 - 3) pos = hit->height * hit->width * 3 - 3;
			return vec3(
				hit->cols[pos],
				hit->cols[pos + 1],
				hit->cols[pos + 2]
			);
		}
		else if ((abs(ray.y) >= abs(ray.x)) && (abs(ray.y) >= abs(ray.z))) {
			if (ray.y > 0.0f) hit = device_py;
			else hit = device_ny;
			int pos = ((int)((ray.z / ray.y + 1.0f) / 2.0f * hit->height)) * 3 * hit->width
				+ ((int)((ray.x / abs(ray.y) + 1.0f) / 2.0f * hit->width)) * 3;
			if (pos > hit->height * hit->width * 3 - 3) pos = hit->height * hit->width * 3 - 3;
			return vec3(
				hit->cols[pos],
				hit->cols[pos + 1],
				hit->cols[pos + 2]
			);
		}
		else if ((abs(ray.z) >= abs(ray.x)) && (abs(ray.z) >= abs(ray.y))) {
			if (ray.z > 0.0f) hit = device_pz;
			else hit = device_nz;
			int pos = ((int)((1.0f - ((ray.y / abs(ray.z) + 1.0f) / 2.0f)) * hit->height)) * 3 * hit->width
				+ ((int)((ray.x / ray.z + 1.0f) / 2.0f * hit->width)) * 3;
			if (pos > hit->height * hit->width * 3 - 3) pos = hit->height * hit->width * 3 - 3;
			return vec3(
				hit->cols[pos],
				hit->cols[pos + 1],
				hit->cols[pos + 2]
			);
		}
	}
};

// The virtual camera
class Camera {
	vec3 eye, lookat, right, up;
public:
	void set(vec3 _eye, vec3 _lookat, vec3 vup, float fov) {
		eye = _eye;
		lookat = _lookat;
		vec3 w = eye - lookat;
		float f = w.length();
		right = cross(vup, w).normalize() * f * tanf(fov / 2.0f);	// orthogonalization
		up = cross(w, right).normalize() * f * tanf(fov / 2.0f);
	}
	__host__ __device__ Ray getRay(float X, float Y) {	// integer parts of X, Y define the pixel, fractional parts the point inside pixel
		vec3 dir = lookat + right * (2.0f * X / screenWidth - 1.0f) + up * (2.0f * Y / screenHeight - 1.0f) - eye;
		return Ray(eye, dir);
	}
};

// Point light source
struct Light {
	vec3 location;
	vec3 power;

	Light() {
	
	}

	Light(vec3 _location, vec3 _power) {
		location = _location;
		power = _power;
	}
	__host__ __device__ float distanceOf(vec3 point) {
		return (location - point).length();
	}
	__host__ __device__ vec3 directionOf(vec3 point) {
		return (location - point).normalize();
	}
	__host__ __device__ vec3 radianceAt(vec3 point) {
		float distance2 = dot(location - point, location - point);
		if (distance2 < epsilon) distance2 = epsilon;
		return power / distance2 / 4.0f / M_PI;
	}
	__device__  void randomSampleRay(vec3& outDir, hiprandState_t* state) {
		vec3 X = vec3(0.0f, 0.0f, 1.0f);
		vec3 Y = vec3(0.0f, 1.0f, 0.0f);
		vec3 Z = vec3(1.0f, 0.0f, 0.0f);

		float alpha = Rand::random(state) * 2.0f * PI;
		float beta = Rand::random(state) * 2.0f * PI;

		outDir = ((X * cosf(alpha) + Z * sinf(alpha)) * sinf(beta) + Y * cosf(beta)).normalize();

		//float x, y, z;
		//do {
		//	x = 2 * Rand::random(state) - 1;    // random point in [-1,-1;1,1]
		//	y = 2 * Rand::random(state) - 1;
		//} while (x * x + y * y > 1);  // reject if not in circle
		//z = sqrtf(1 - x * x - y * y);  // project to hemisphere

		//outDir = X * z + Y * x + Z * y;
	}
};

// Virtual world
class Scene {
	Sphere* device_spheres;
	int* device_sphere_size;

	MeshObject* device_meshes;
	int* device_meshes_size;

	Light* device_lights;
	int* device_lights_size;

	Plane* device_planes;
	int* device_plane_size;

	Ring* device_rings;
	int* device_ring_size;

public:
	Camera camera;
	EnvMap envMap = EnvMap("resources/hdr-env-cube/");
	void build() {
		vec3 eye = vec3(0.0f, 0.0f, 3.0f);
		vec3 vup = vec3(0.0f, 1.0f, 0.0f);
		vec3 lookat = vec3(0.0f, 0.0f, 0.0f);
		float fov = 100.0f * M_PI / 180.0f;
		camera.set(eye, lookat, vup, fov);

		//LIGHTS
		int light_size = 1;
		copyToDevice(device_lights_size, light_size, "device_lights_size");

		Light* lights = new Light[light_size]; 
		lights[0] = Light(vec3(0.0f, -4.0f, -4.5f), vec3(1000.0f, 1000.0f, 1000.0f));
		//lights[1] = Light(vec3(0, 10, -3), vec3(2000, 2000, 2000));
		//lights[1] = Light(vec3(0, 6, 4), vec3(2000, 2000, 2000));
		//lights[2] = Light(vec3(0, 2, -2), vec3(2000, 2000, 2000));
		//lights[3] = Light(vec3(0, 6, 2), vec3(2000, 2000, 2000));

		copyToDevice(device_lights, lights, light_size, "device_lights");

		delete lights;

		//SPHERE
		int spheres_size = 0;
		copyToDevice(device_sphere_size, spheres_size, "device_sphere_size");

		Sphere* spheres = new Sphere[spheres_size];
		//spheres[0] = Sphere(vec3(0, 0, 0), 10, new Material(vec3(0.2,0.9,0.2), vec3(0,0,0)));
		//spheres[1] = Sphere(vec3(-1.5, 0, 0), 0.6, new Material(vec3(0, 0, 0), vec3(1, 1, 1)));

		copyToDevice(device_spheres, spheres, spheres_size, "device_spheres");

		delete spheres;

		//PLANES
		int planes_size = 6;
		copyToDevice(device_plane_size, planes_size, "device_plane_size");

		Plane* planes = new Plane[planes_size];
		planes[0] = Plane(vec3(0.0f, -5.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[1] = Plane(vec3(0.0f, 0.0f, 5.0f), vec3(0.0f, 0.0f, 1.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[2] = Plane(vec3(0.0f, 0.0f, -5.0f), vec3(0.0f, 0.0f, 1.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[3] = Plane(vec3(5.0f, 0.0f, 0.0f), vec3(1.0f, 0.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[4] = Plane(vec3(-5.0f, 0.0f, 0.0f), vec3(1.0f, 0.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[5] = Plane(vec3(0.0f, 5.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));

		copyToDevice(device_planes, planes, planes_size, "device_planes");

		delete planes;

		//RINGS
		int rings_size = 0;
		copyToDevice(device_ring_size, rings_size, "device_plane_size");

		Ring* rings = new Ring[rings_size];
		//rings[0] = Ring(
		//	vec3(0, -0.5, 0), 
		//	vec3(0, 1, 0),
		//	3,
		//	1.5,
		//	new Material(vec3(0.0, 0.0, 0.0), vec3(0.9, 0.9, 0.9))
		//);

		copyToDevice(device_rings, rings, rings_size, "device_rings");

		delete rings;

		//MESHES
		int mesh_size = 3;
		copyToDevice(device_meshes_size, mesh_size, "device_meshes_size");

		MeshObject* meshes = new MeshObject[mesh_size];

		//meshes[0] = MeshObject(
		//	"resources/objects/bunny.obj",
		//	vec3(0.3, 0, 0), //position
		//	vec3(-3.1415/ 2.0, 0.8, 0.0),  //rotate
		//	vec3(0.5, 0.5, 0.5),  //scale
		//	new Material(vec3(0.0, 0.0, 0.0), vec3(0.9, 0.9, 0.9))
		//);

		meshes[0] = MeshObject(
			"resources/objects/cube.obj",
			vec3(2.0f, -4.0f, -4.0f), //position
			vec3(0.0f, 0.0f, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.8392f, 0.0f, 0.4392f), vec3(0.0f, 0.0f, 0.0f))
		);

		meshes[1] = MeshObject(
			"resources/objects/cube.obj",
			vec3(0.0f, -4.0f, -2.5f), //position
			vec3(0.0f, 0.0f, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.6078f, 0.3098f, 0.5882f), vec3(0.0f, 0.0f, 0.0f))
		);
		meshes[2] = MeshObject(
			"resources/objects/cube.obj",
			vec3(-2.0f, -4.0f, -4.0f), //position
			vec3(0.0f, M_PI, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.0f, 0.2196f, 0.6588f), vec3(0.0f, 0.0f, 0.0f))
		);

		copyToDevice(device_meshes, meshes, mesh_size, "device_meshes");

		delete meshes;
	}

	// Find the first intersection of the ray with objects
	__device__ Hit firstIntersect(Ray ray) {
		Hit bestHit;
		for (int i = 0; i < *device_meshes_size; i++) {
			Hit hit = device_meshes[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		for (int i = 0; i < *device_sphere_size; i++) {
			Hit hit = device_spheres[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		for (int i = 0; i < *device_plane_size; i++) {
			Hit hit = device_planes[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		for (int i = 0; i < *device_ring_size; i++) {
			Hit hit = device_rings[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		return bestHit;
	}

	__device__ float clamp(float x, float min, float max) {
		if (x < min) {
			return min;
		}
		if (x > max) {
			return max;
		}
		return x;
	}

	// Trace a ray and return the radiance of the visible surface
	__device__ vec3 trace(Ray _ray, EnvMap envMap, hiprandState_t* state) {
		vec3 outRad(0.0f, 0.0f, 0.0f);

		//random light ray
		int light_idx = (int)(Rand::random(state) * ((float)*device_lights_size));
		vec3 light_source_out_dir;
		device_lights[light_idx].randomSampleRay(light_source_out_dir, state);

		//first hit
		Hit first_light_hit = firstIntersect(Ray(device_lights[light_idx].location, light_source_out_dir));
		if (first_light_hit.t < 0.0f) {
			return outRad;
		}

		Hit light_paths_hits[maxdepth];
		vec3 light_paths_radiance[maxdepth];
		float light_paths_pdf[maxdepth];
		light_paths_hits[0] = first_light_hit;
		light_paths_radiance[0] = device_lights[light_idx].radianceAt(first_light_hit.position);
		light_paths_pdf[0] = 1.0f / ((float)*device_lights_size) * 1.0f / (4.0f * M_PI);
		
		//make light paths
		for (int i = 0; i < maxdepth; i++) {

			float diffuseSelectProb = light_paths_hits[i].material->diffuseAlbedo.average();
			float mirrorSelectProb = light_paths_hits[i].material->mirrorAlbedo.average();
			float rnd = Rand::random(state);	// Russian roulette to find diffuse, mirror or no reflection

			if (rnd < diffuseSelectProb + mirrorSelectProb) {
				vec3 lightDirIn = (i == 0) ?
					light_source_out_dir :
					(light_paths_hits[i].position - light_paths_hits[i - 1].position).normalize();
				vec3 lightDirOut;
				float pdf_brdf;

				if (rnd < diffuseSelectProb) { // diffuse
					pdf_brdf = SampleDiffuse(
						light_paths_hits[i].normal,
						lightDirIn,
						lightDirOut,
						state
					) * diffuseSelectProb;
					float cosThetaL = dot(light_paths_hits[i].normal, lightDirIn * (-1));
					if (cosThetaL > epsilon) {
						if (i == 0) {
							light_paths_radiance[i] = light_paths_radiance[i] * (light_paths_hits[i].material->diffuseAlbedo) / M_PI * cosThetaL;
						}
						else {
							light_paths_radiance[i] = light_paths_radiance[i - 1] * (light_paths_hits[i].material->diffuseAlbedo) / M_PI * cosThetaL;
						}
					}
					else {
						light_paths_hits[i].t = -1;
						break;
					}
				}

				else { // mirror
					pdf_brdf = SampleMirror(
						light_paths_hits[i].normal,
						lightDirIn,
						lightDirOut
					)* mirrorSelectProb;
					float cosThetaL = dot(light_paths_hits[i].normal, lightDirIn * (-1));
					if (cosThetaL > epsilon) {
						if (i == 0) {
							light_paths_radiance[i] = light_paths_radiance[i] * (light_paths_hits[i].material->mirrorAlbedo);
						}
						else {
							light_paths_radiance[i] = light_paths_radiance[i - 1] * (light_paths_hits[i].material->mirrorAlbedo);
						}
					}
					else {
						light_paths_hits[i].t = -1;
						break;
					}
				}

				//evaluate next hit
				if (i != maxdepth - 1) {
					light_paths_hits[i + 1] = firstIntersect(Ray(light_paths_hits[i].position + light_paths_hits[i].normal * epsilon, lightDirOut));
					if (light_paths_hits[i + 1].t < 0) {
						break;
					}
					else {
						light_paths_pdf[i + 1] = light_paths_pdf[i] * pdf_brdf;
					}
				}
			}
			else {
				break;
			}
		}


		vec3 paths_color[maxdepth * (maxdepth + 1)];
		float paths_probability[maxdepth * (maxdepth + 1)];
		int n_paths = 0;


		vec3 pixel_path_brdf = vec3(1.0f, 1.0f, 1.0f);
		float pixel_path_pdf = 1.0f;
		Ray ray = _ray;

		//make paths
		for (int i = 0; i < maxdepth; i++) {
			Hit hit = firstIntersect(ray);
			if (hit.t < 0) {
				//outRad += envMap.getPixelColor(ray.dir) * 6* pixel_path_weight;
				break;
				return outRad;
			}
			else {

				//for (int i = 0; i < *device_lights_size; i++) {	// Direct light source computation
				// vec3 outDir = device_lights[i].directionOf(hit.position);
				//	Hit shadowHit = firstIntersect(Ray(hit.position + hit.normal * epsilon, outDir));
				//	if (shadowHit.t < epsilon || shadowHit.t > device_lights[i].distanceOf(hit.position)) {	// if not in shadow
				//		float cosThetaL = dot(hit.normal, outDir);
				//		if (cosThetaL >= epsilon) {
				//			outRad += hit.material->diffuseAlbedo / M_PI * cosThetaL
				//				* device_lights[i].radianceAt(hit.position) * pixel_path_brdf /pixel_path_pdf;
				//		}
				//	}
				//}


				vec3 outDir_light = device_lights[light_idx].directionOf(hit.position);
				Hit shadowHit = firstIntersect(Ray(hit.position + hit.normal * epsilon, outDir_light));
				if (shadowHit.t < epsilon || shadowHit.t > device_lights[light_idx].distanceOf(hit.position)) {	// if not in shadow
					float cosThetaL = dot(hit.normal, outDir_light);
					if (cosThetaL >= epsilon) {
						paths_color[n_paths] = device_lights[light_idx].radianceAt(hit.position)
							* hit.material->diffuseAlbedo / M_PI * cosThetaL
							* pixel_path_brdf
							;
						paths_probability[n_paths] = pixel_path_pdf;
						n_paths++;
					}
				}


				for (int j = 0; j<maxdepth; j++) {
					if (light_paths_hits[j].t < 0) break;

					Ray pixel_path_to_light_path = Ray(
						hit.position + hit.normal * epsilon,
						(light_paths_hits[j].position - hit.position).normalize()
					);

					Hit pixel_to_light_first_hit = firstIntersect(pixel_path_to_light_path);
					bool clearView = pixel_to_light_first_hit.t > 0 && (pixel_to_light_first_hit.position - light_paths_hits[i].position).length() < epsilon;

					if (clearView) {
						float r = (light_paths_hits[j].position - hit.position).length();
						float cosThetaInPixel = dot(hit.normal, pixel_path_to_light_path.dir);
						float cosThetaInLight = dot(light_paths_hits[j].normal, pixel_path_to_light_path.dir * (-1.0));
						if (cosThetaInPixel < epsilon) {
							continue;
						}
						vec3 brdf_pixel = (hit.material->diffuseAlbedo) / M_PI * cosThetaInPixel;

						paths_color[n_paths] = light_paths_radiance[j]
							* brdf_pixel
							* cosThetaInPixel
							//* 1.0 / (r * r)
							* pixel_path_brdf
							;
						paths_probability[n_paths] = 1;// pixel_path_pdf* light_paths_pdf[j];
						n_paths++;
					}

				}

				float diffuseSelectProb = hit.material->diffuseAlbedo.average();
				float mirrorSelectProb = hit.material->mirrorAlbedo.average();

				float rnd = Rand::random(state);	// Russian roulette to find diffuse, mirror or no reflection
				vec3 outDir;
				if (rnd < diffuseSelectProb) { // diffuse
					float pdf = SampleDiffuse(hit.normal, ray.dir, outDir, state);
					float cosThetaL = dot(hit.normal, outDir);
					if (cosThetaL >= epsilon) {
						pixel_path_brdf = pixel_path_brdf * (hit.material->diffuseAlbedo) / M_PI * cosThetaL; //brdf
						pixel_path_pdf = pixel_path_pdf * pdf * diffuseSelectProb; //pdf
					}
					else {
						break;
					}
				}
				else if (rnd < diffuseSelectProb + mirrorSelectProb) { // mirror
					float pdf = SampleMirror(hit.normal, ray.dir, outDir);
					pixel_path_brdf = pixel_path_brdf * hit.material->mirrorAlbedo; //brdf
					pixel_path_pdf = pixel_path_pdf * pdf * mirrorSelectProb; //pdf
				}
				else {
					break;
				}

				ray = Ray(hit.position + hit.normal * epsilon, outDir);
			}
		}

		if (n_paths != 0) {
			float sum_probability = 0;
			for (int i = 0; i < n_paths; i++) {
				sum_probability += paths_probability[i];
			}
			//return vec3(sum_probability, sum_probability, sum_probability);
			for (int i = 0; i < n_paths; i++) {
				outRad += paths_color[i] / paths_probability[i];// / n_paths;// sum_probability;
			}
		}
		return outRad;
	}
};

// Render the scene: Trace nSamples rays through each pixel and average radiance values
__global__ void render(vec3* image, Scene scene) {
	//indicies
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\rProgress: %d%%", (int)(((float)id / (screenHeight * screenWidth)) * 100));
	if (id < screenHeight * screenWidth) {
		int tileIdx = id / tileArea;

		int tile_row = tileIdx / (screenWidth / tileSize);
		int tile_column = tileIdx - (tile_row * (screenWidth / tileSize));

		int offest_on_tile = id - (tileIdx * tileArea);
		int row_offset = offest_on_tile / tileSize;
		int column_offset = offest_on_tile - (row_offset * tileSize);

		int row = tile_row * tileSize + row_offset;
		int column = tile_column * tileSize + column_offset;

		id = row * screenWidth + column;

		//init rand
		hiprandState_t state;
		hiprand_init(id, /* the seed controls the sequence of random values that are produced */
			0, /* the sequence number is only important with multiple cores */
			1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&state);

		//fill image
		image[id] = vec3(0.0f, 0.0f, 0.0f);
		for (int i = 0; i < nSamples; i++) {
			image[id] += scene.trace(
				scene.camera.getRay(column + Rand::random(&state), row + Rand::random(&state)),
				scene.envMap,
				&state
			) / nSamples;
		}
	}
}

// Save image into a Targa format file
void SaveTGAFile(char* fileName, const vec3* image) {
	FILE* tgaFile = fopen(fileName, "wb");
	if (!tgaFile) {
		printf("File %s cannot be opened\n", fileName);
		return;
	}
	// File header
	fputc(0, tgaFile); fputc(0, tgaFile); fputc(2, tgaFile);
	for (int i = 3; i < 12; i++) { fputc(0, tgaFile); }
	fputc(screenWidth % 256, tgaFile); fputc(screenWidth / 256, tgaFile);
	fputc(screenHeight % 256, tgaFile); fputc(screenHeight / 256, tgaFile);
	fputc(24, tgaFile); fputc(32, tgaFile);
	// List of pixel colors
	for (int Y = screenHeight - 1; Y >= 0; Y--) {
		for (int X = 0; X < screenWidth; X++) {
			int R = (int)fmaxf(fminf(image[Y * screenWidth + X].x * 255.5f, 255.5f), 0.0f);
			int G = (int)fmaxf(fminf(image[Y * screenWidth + X].y * 255.5f, 255.5f), 0.0f);
			int B = (int)fmaxf(fminf(image[Y * screenWidth + X].z * 255.5f, 255.5f), 0.0f);
			fputc(B, tgaFile); fputc(G, tgaFile); fputc(R, tgaFile);
		}
	}
	fclose(tgaFile);
}


int main(int argc, char* argv[]) {
	//cuda setDevice
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	//Scene
	Scene scene;										
	scene.build();

	//Malloc image on GPU
	vec3* device_image;
	cudaStatus = hipMalloc((void**)&device_image, screenHeight * screenWidth * sizeof(vec3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	//Render scene to image on GPU
	std::cout <<"Render strated" << std::endl;
	auto begin = std::chrono::high_resolution_clock::now();

	render<<<screenWidth * screenHeight / tileArea + 1, tileArea>>>(device_image, scene);

	cudaStatus = hipGetLastError();
	//check for errors
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "render launch failed! %s", hipGetErrorString(cudaStatus));
	}
	//sync
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching render!\n", cudaStatus);
	}

	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
	std::cout << "Render ended in " << elapsed.count() * 1e-9 << std::endl;

	//copy image to CPU
	vec3* image = new vec3[screenHeight * screenWidth];
	cudaStatus = hipMemcpy(image, device_image, screenHeight * screenWidth * sizeof(vec3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "device_image hipMemcpy failed!");
	}

	//save image
	char location[30] = "resources/out/image.tga";		
	SaveTGAFile(location, image);		
	
	hipDeviceReset();
	//delete image
	delete image;
	return 1;
}