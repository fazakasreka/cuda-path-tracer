#include "hip/hip_runtime.h"
﻿//=============================================================================================
// Path tracing program
//=============================================================================================
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <math.h>
#include <vector>
#include <string>
#include <iostream>
#include <chrono>
#include <queue> 

#include "copyToDevice/copyToDevice.h"
#include "HDREnvMap/envMap.cuh"
#include "math/math.cuh"
#include "constants/constants.cuh"

#include "pathTracing/camera.cuh"
#include "pathTracing/hit.cuh"
#include "pathTracing/intersectable.cuh"
#include "pathTracing/light.cuh"
#include "pathTracing/material.cuh"
#include "pathTracing/random.cuh"
#include "pathTracing/ray.cuh"
#include "pathTracing/samplers.cuh"

#include "shapes/mesh.cuh"
#include "shapes/sphere.cuh"
#include "shapes/plane.cuh"

class Scene {
	Sphere* device_spheres;
	int* device_sphere_size;

	MeshObject* device_meshes;
	int* device_meshes_size;

	Light* device_lights;
	int* device_lights_size;

	Plane* device_planes;
	int* device_plane_size;

public:
	Camera camera;
	EnvMap envMap = EnvMap("resources/hdr-env-cube/");
	void build() {
		vec3 eye = vec3(0.0f, 0.0f, 3.0f);
		vec3 vup = vec3(0.0f, 1.0f, 0.0f);
		vec3 lookat = vec3(0.0f, 0.0f, 0.0f);
		float fov = 100.0f * PI / 180.0f;
		camera.set(eye, lookat, vup, fov);

		//LIGHTS
		int light_size = 1;
		copyToDevice(device_lights_size, light_size, "device_lights_size");

		Light* lights = new Light[light_size]; 
		lights[0] = Light(vec3(0.0f, -4.0f, -4.5f), vec3(1000.0f, 1000.0f, 1000.0f));
		//lights[1] = Light(vec3(0, 10, -3), vec3(2000, 2000, 2000));
		//lights[1] = Light(vec3(0, 6, 4), vec3(2000, 2000, 2000));
		//lights[2] = Light(vec3(0, 2, -2), vec3(2000, 2000, 2000));
		//lights[3] = Light(vec3(0, 6, 2), vec3(2000, 2000, 2000));

		copyToDevice(device_lights, lights, light_size, "device_lights");

		delete lights;

		//SPHERE
		int spheres_size = 0;
		copyToDevice(device_sphere_size, spheres_size, "device_sphere_size");

		Sphere* spheres = new Sphere[spheres_size];
		//spheres[0] = Sphere(vec3(0, 0, 0), 10, new Material(vec3(0.2,0.9,0.2), vec3(0,0,0)));
		//spheres[1] = Sphere(vec3(-1.5, 0, 0), 0.6, new Material(vec3(0, 0, 0), vec3(1, 1, 1)));

		copyToDevice(device_spheres, spheres, spheres_size, "device_spheres");

		delete spheres;

		//PLANES
		int planes_size = 6;
		copyToDevice(device_plane_size, planes_size, "device_plane_size");

		Plane* planes = new Plane[planes_size];
		planes[0] = Plane(vec3(0.0f, -5.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[1] = Plane(vec3(0.0f, 0.0f, 5.0f), vec3(0.0f, 0.0f, 1.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[2] = Plane(vec3(0.0f, 0.0f, -5.0f), vec3(0.0f, 0.0f, 1.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[3] = Plane(vec3(5.0f, 0.0f, 0.0f), vec3(1.0f, 0.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[4] = Plane(vec3(-5.0f, 0.0f, 0.0f), vec3(1.0f, 0.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));
		planes[5] = Plane(vec3(0.0f, 5.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), new Material(vec3(0.9f, 0.9f, 0.9f), vec3(0.0f, 0.0f, 0.0f)));

		copyToDevice(device_planes, planes, planes_size, "device_planes");

		delete planes;

		//MESHES
		int mesh_size = 3;
		copyToDevice(device_meshes_size, mesh_size, "device_meshes_size");

		MeshObject* meshes = new MeshObject[mesh_size];

		//meshes[0] = MeshObject(
		//	"resources/objects/bunny.obj",
		//	vec3(0.3, 0, 0), //position
		//	vec3(-3.1415/ 2.0, 0.8, 0.0),  //rotate
		//	vec3(0.5, 0.5, 0.5),  //scale
		//	new Material(vec3(0.0, 0.0, 0.0), vec3(0.9, 0.9, 0.9))
		//);

		meshes[0] = MeshObject(
			"resources/objects/cube.obj",
			vec3(2.0f, -4.0f, -4.0f), //position
			vec3(0.0f, 0.0f, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.8392f, 0.0f, 0.4392f), vec3(0.0f, 0.0f, 0.0f))
		);

		meshes[1] = MeshObject(
			"resources/objects/cube.obj",
			vec3(0.0f, -4.0f, -2.5f), //position
			vec3(0.0f, 0.0f, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.6078f, 0.3098f, 0.5882f), vec3(0.0f, 0.0f, 0.0f))
		);
		meshes[2] = MeshObject(
			"resources/objects/cube.obj",
			vec3(-2.0f, -4.0f, -4.0f), //position
			vec3(0.0f, PI, 0.0f),  //rotate
			vec3(1.0f, 1.0f, 1.0f),  //scale
			new Material(vec3(0.0f, 0.2196f, 0.6588f), vec3(0.0f, 0.0f, 0.0f))
		);

		copyToDevice(device_meshes, meshes, mesh_size, "device_meshes");

		delete meshes;
	}

	// Find the first intersection of the ray with objects
	__device__ Hit firstIntersect(Ray ray) {
		Hit bestHit;
		for (int i = 0; i < *device_meshes_size; i++) {
			Hit hit = device_meshes[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		for (int i = 0; i < *device_sphere_size; i++) {
			Hit hit = device_spheres[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		for (int i = 0; i < *device_plane_size; i++) {
			Hit hit = device_planes[i].intersect(ray); //  hit.t < 0 if no intersection
			if (hit.t > 0 && (bestHit.t < 0 || hit.t < bestHit.t)) bestHit = hit;
		}
		return bestHit;
	}

	__device__ float clamp(float x, float min, float max) {
		if (x < min) {
			return min;
		}
		if (x > max) {
			return max;
		}
		return x;
	}

	// Bidirectional Path Tracer
	__device__ vec3 trace(Ray _ray, EnvMap envMap, hiprandState_t* state) {
		vec3 outRad(0.0f, 0.0f, 0.0f);
		//LIGHT PATH

		//choose a random light source
		int light_idx = (int)(Rand::random(state) * ((float)*device_lights_size));
		vec3 light_out_dir;
		device_lights[light_idx].randomSampleRay(light_out_dir, state);

		//get the first object it hits
		Ray light_ray(device_lights[light_idx].location, light_out_dir);
		Hit hit = firstIntersect(light_ray);
		if (hit.t < 0) return outRad;

		//store the hits and throughputs
		Hit light_hits[maxdepth];
		vec3 light_throughput[maxdepth];
		float light_pdf[maxdepth];

		//initialize the first item
		light_hits[0] = hit;
		light_throughput[0] = device_lights[light_idx].radianceAt(hit.position);
		light_pdf[0] = (1.0f / (float)*device_lights_size) * (1.0f / (4.0f * PI));

		for (int i = 0; i < maxdepth - 1; i++) {
			float diffProb = light_hits[i].material->diffuseAlbedo.average();
			float mirrorProb = light_hits[i].material->mirrorAlbedo.average();
			float rnd = Rand::random(state);

			if (rnd >= diffProb + mirrorProb) break;

			vec3 inDir = (i == 0) ? light_out_dir :
				(light_hits[i].position - light_hits[i - 1].position).normalize();
			vec3 outDir;
			float pdf_brdf;

			if (rnd < diffProb) {
				pdf_brdf = SampleDiffuse(light_hits[i].normal, inDir, outDir, state) * diffProb;
				float cosTheta = dot(light_hits[i].normal, -inDir);
				if (cosTheta < epsilon) break;
				light_throughput[i] = (i == 0) 
					? light_throughput[i] * light_hits[i].material->diffuseAlbedo / PI * cosTheta
					: light_throughput[i - 1] * light_hits[i].material->diffuseAlbedo / PI * cosTheta;
			} else {
				pdf_brdf = SampleMirror(light_hits[i].normal, inDir, outDir) * mirrorProb;
				float cosTheta = dot(light_hits[i].normal, -inDir);
				if (cosTheta < epsilon) break;
				light_throughput[i] = (i == 0) 
					? light_throughput[i] * light_hits[i].material->mirrorAlbedo
					: light_throughput[i - 1] * light_hits[i].material->mirrorAlbedo;
			}

			Ray bounce_ray(light_hits[i].position + light_hits[i].normal * epsilon, outDir);
			Hit next_hit = firstIntersect(bounce_ray);
			if (next_hit.t < 0) break;

			light_hits[i + 1] = next_hit;
			light_pdf[i + 1] = light_pdf[i] * pdf_brdf;
		}

		// CAMERA PATH

		vec3 cam_throughput = vec3(1.0f, 1.0f, 1.0f);
		float cam_pdf = 1.0f;
		Ray cam_ray = _ray;

		for (int i = 0; i < maxdepth; i++) {
			Hit hit = firstIntersect(cam_ray);
			if (hit.t < 0) break;

			//connect current path with the light source
			vec3 light_pos = device_lights[0].location;
			vec3 light_dir = (hit.position - light_pos).normalize();
			float dist = (light_pos - hit.position).length();

			Hit shadow = firstIntersect(Ray(light_pos, light_dir));
			if (shadow.t > 0 && (shadow.position - hit.position).length() < epsilon) {
				vec3 light_radiance = device_lights[0].radianceAt(hit.position);
				float cosTheta = dot(hit.normal, -light_dir);
				if (cosTheta > epsilon) {
					vec3 brdf = hit.material->diffuseAlbedo / PI;
					vec3 contrib = (cam_throughput / cam_pdf) * (light_radiance * brdf * cosTheta);
					outRad += contrib;
				}
			}

			//connect current path with every element of the light path
			for (int j = 0; j < maxdepth; j++) {
				if (light_hits[j].t < 0) break;

				vec3 dir = (light_hits[j].position - hit.position).normalize();
				float dist = (light_hits[j].position - hit.position).length();

				//check if they can be connected
				Hit shadow = firstIntersect(Ray(hit.position + hit.normal * epsilon, dir));
				if (shadow.t < 0 || (shadow.position - light_hits[j].position).length() > epsilon) continue;

				float cosCam = dot(hit.normal, dir);
				float cosLight = dot(light_hits[j].normal, -dir);
				if (cosCam < epsilon || cosLight < epsilon) continue;

				float G = cosCam * cosLight / (dist * dist);
				vec3 brdf = hit.material->diffuseAlbedo / PI;
				vec3 camera_weight = cam_throughput * brdf * cosCam / cam_pdf;
				vec3 light_weight = light_throughput[j] / light_pdf[j];

				outRad += light_weight * G * camera_weight;
			}

			//get next element in camera path
			float diffProb = hit.material->diffuseAlbedo.average();
			float mirrorProb = hit.material->mirrorAlbedo.average();
			float rnd = Rand::random(state);
			vec3 outDir;

			if (rnd < diffProb) {
				float pdf = SampleDiffuse(hit.normal, cam_ray.dir, outDir, state);
				float cosTheta = dot(hit.normal, outDir);
				if (cosTheta < epsilon) break;
				cam_throughput = cam_throughput * hit.material->diffuseAlbedo / PI * cosTheta;
				cam_pdf = cam_pdf * pdf * diffProb;
			} else if (rnd < diffProb + mirrorProb) {
				float pdf = SampleMirror(hit.normal, cam_ray.dir, outDir);
				cam_throughput = cam_throughput * hit.material->mirrorAlbedo;
				cam_pdf = cam_pdf * pdf * mirrorProb;
			} else {
				break;
			}

			cam_ray = Ray(hit.position + hit.normal * epsilon, outDir);
		}

		return outRad;
	}
};

// Render the scene: Trace nSamples rays through each pixel and average radiance values
__global__ void render(vec3* image, Scene scene) {
	//indicies
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\rProgress: %d%%", (int)(((float)id / (screenHeight * screenWidth)) * 100));
	if (id < screenHeight * screenWidth) {
		int tileIdx = id / tileArea;

		int tile_row = tileIdx / (screenWidth / tileSize);
		int tile_column = tileIdx - (tile_row * (screenWidth / tileSize));

		int offest_on_tile = id - (tileIdx * tileArea);
		int row_offset = offest_on_tile / tileSize;
		int column_offset = offest_on_tile - (row_offset * tileSize);

		int row = tile_row * tileSize + row_offset;
		int column = tile_column * tileSize + column_offset;

		id = row * screenWidth + column;

		//init rand
		hiprandState_t state;
		hiprand_init(id, /* the seed controls the sequence of random values that are produced */
			0, /* the sequence number is only important with multiple cores */
			1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&state);

		//fill image
		image[id] = vec3(0.0f, 0.0f, 0.0f);
		for (int i = 0; i < nSamples; i++) {
			image[id] += scene.trace(
				scene.camera.getRay(column + Rand::random(&state), row + Rand::random(&state)),
				scene.envMap,
				&state
			) / nSamples;
		}
	}
}

// Save image into a Targa format file
void SaveTGAFile(char* fileName, const vec3* image) {
	FILE* tgaFile = fopen(fileName, "wb");
	if (!tgaFile) {
		printf("File %s cannot be opened\n", fileName);
		return;
	}
	// File header
	fputc(0, tgaFile); fputc(0, tgaFile); fputc(2, tgaFile);
	for (int i = 3; i < 12; i++) { fputc(0, tgaFile); }
	fputc(screenWidth % 256, tgaFile); fputc(screenWidth / 256, tgaFile);
	fputc(screenHeight % 256, tgaFile); fputc(screenHeight / 256, tgaFile);
	fputc(24, tgaFile); fputc(32, tgaFile);
	// List of pixel colors
	for (int Y = screenHeight - 1; Y >= 0; Y--) {
		for (int X = 0; X < screenWidth; X++) {
			int R = (int)fmaxf(fminf(image[Y * screenWidth + X].x * 255.5f, 255.5f), 0.0f);
			int G = (int)fmaxf(fminf(image[Y * screenWidth + X].y * 255.5f, 255.5f), 0.0f);
			int B = (int)fmaxf(fminf(image[Y * screenWidth + X].z * 255.5f, 255.5f), 0.0f);
			fputc(B, tgaFile); fputc(G, tgaFile); fputc(R, tgaFile);
		}
	}
	fclose(tgaFile);
}


int main(int argc, char* argv[]) {
	//cuda setDevice
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	//Scene
	Scene scene;										
	scene.build();

	//Malloc image on GPU
	vec3* device_image;
	cudaStatus = hipMalloc((void**)&device_image, screenHeight * screenWidth * sizeof(vec3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	//Render scene to image on GPU
	std::cout <<"Render strated" << std::endl;
	auto begin = std::chrono::high_resolution_clock::now();

	render<<<screenWidth * screenHeight / tileArea + 1, tileArea>>>(device_image, scene);

	cudaStatus = hipGetLastError();
	//check for errors
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "render launch failed! %s", hipGetErrorString(cudaStatus));
	}
	//sync
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching render!\n", cudaStatus);
	}

	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
	std::cout << "\nRender ended in " << elapsed.count() * 1e-9 << std::endl;

	//copy image to CPU
	vec3* image = new vec3[screenHeight * screenWidth];
	cudaStatus = hipMemcpy(image, device_image, screenHeight * screenWidth * sizeof(vec3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "device_image hipMemcpy failed!");
	}

	//save image
	char location[30] = "resources/out/image.tga";		
	SaveTGAFile(location, image);		
	
	hipDeviceReset();

	//delete image
	
	delete image;
	return 1;
}